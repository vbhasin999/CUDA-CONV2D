#include "hip/hip_runtime.h"
#include "conv2d_impl.h"
#include <stdio.h>

#define BLOCK_DIM_X 2
#define BLOCK_DIM_Y 2
#define BLOCK_DIM_Z 1
#define PIXELS_PER_THREAD 16

// Add batch, stride and padding later
// we could have 1 block <-> 1 output channel
// and 1 thread <-> 1 output pixel

template <typename T>
__global__ void conv_kernel_basic(
    T* __restrict__ result, 
    const T* __restrict__ input, 
    const T* __restrict__ filter, 
    int Cin, int H, int W, int Cout, int K) {

    int out_x = threadIdx.x + blockIdx.x * blockDim.x;  // Output width index
    int out_y = threadIdx.y + blockIdx.y * blockDim.y;  // Output height index
    int out_c = blockIdx.z;                            // Output channel index                 

    int H_out = H - K + 1;
    int W_out = W - K + 1;

    // Shared memory for reduction
    __shared__ T partial_sums[BLOCK_DIM_X][BLOCK_DIM_Y];  
    // Initialize shared memory only once per block
    if (threadIdx.z == 0) partial_sums[threadIdx.x][threadIdx.y]= 0.0f;

    __syncthreads();

    T local_sum = 0;
    if (out_x < H_out && out_y < W_out) {
        // Loop over input channels in chunks
        for (int in_c = threadIdx.z; in_c < Cin; in_c += BLOCK_DIM_Z) {
            for (int kx = 0; kx < K; ++kx) { // Kernel rows
                for (int ky = 0; ky < K; ++ky) { // Kernel columns
                    int in_x = out_x + kx;
                    int in_y = out_y + ky;

                    if (in_x < W && in_y < H) {  // Bounds check
                        local_sum += input[in_c * H * W + in_y * W + in_x] *
                                     filter[out_c * Cin * K * K + in_c * K * K + ky * K + kx];
                    }
                }
            }
        }

        // Atomic addition to shared memory
        atomicAdd(&partial_sums[threadIdx.x][threadIdx.y], local_sum);
    }

    __syncthreads();

    // Write final result to global memory
    if (threadIdx.z == 0) {
        if (out_x < H_out && out_y < W_out) {
            result[out_c * H_out * W_out + out_y * W_out + out_x] = partial_sums[threadIdx.x][threadIdx.y];
        }
    }
}


template <typename T>
void launch_conv2d_basic(T *h_result, const T *h_x, const T *h_y, int Cin, int H, int W, int Cout, int K) {
    // Output dimensions
    int H_out = H - K + 1;
    int W_out = W - K + 1;

    // Device memory allocation
    T *d_x, *d_y, *d_result;
    size_t input_size = Cin * H * W * sizeof(T);
    size_t filter_size = Cout * Cin * K * K * sizeof(T);
    size_t output_size = Cout * H_out * W_out * sizeof(T);

    hipMalloc(&d_x, input_size);
    hipMalloc(&d_y, filter_size);
    hipMalloc(&d_result, output_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "1. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Copy inputs to device
    hipMemcpy(d_x, h_x, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, filter_size, hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "2. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z);  
    dim3 gridDim((W_out + blockDim.x - 1) / blockDim.x,
                (H_out + blockDim.y - 1) / blockDim.y,
                Cout);   

    // Launch kernel
    conv_kernel_basic<<<gridDim, blockDim>>>(d_result, d_x, d_y, Cin, H, W, Cout, K);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "3. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Copy results back to host
    hipMemcpy(h_result, d_result, output_size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
}

template <typename T>
__global__ void conv_kernel_opt(
    T* __restrict__ result, 
    const T* __restrict__ input, 
    const T* __restrict__ filter, 
    int Cin, int H, int W, int Cout, int K) {

    int out_x = threadIdx.x + blockIdx.x * blockDim.x;  // Output width index
    int out_y = threadIdx.y + blockIdx.y * blockDim.y;  // Output height index
    int in_c = blockIdx.z;                            // Input channel index                 

    int H_out = H - K + 1;
    int W_out = W - K + 1;

    // should be of size blockDim.x + K - 1 * blockDim.y + K - 1 i.e. 
    // the receptive field of the block
    extern __shared__ T sInput[];
    if ((threadIdx.z == 0) && (threadIdx.x % K == 0) && (threadIdx.y % K == 0)) {
        for (int i = 0; i < K; ++i) {
            for (int j = 0; j < K; ++j) {
                int in_x = out_x + i;
                int in_y = out_y + j;
                
                // printf("Block: (%d, %d, %d), Thread: (%d, %d, %d), loading: %f into smem\n",
                //             blockIdx.x, blockIdx.y, blockIdx.z,
                //             threadIdx.x, threadIdx.y, threadIdx.z,
                //             input[in_c * H * W + in_y * W + in_x]);
                sInput[(threadIdx.y + i) * (BLOCK_DIM_X + K - 1) + threadIdx.x + j] = (in_x < W && in_y < H) ? input[in_c * H * W + in_y * W + in_x] : 0.0f;
            }
        }
    }

    __syncthreads();

    // if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
    //     printf("smem loaded: %f %f %f %f", sInput[0], sInput[1], sInput[2],sInput[3]);
    // }

    
    if (out_x < H_out && out_y < W_out && in_c < Cin) {
        // Loop over output channels in chunks
        for (int out_c = threadIdx.z; out_c < Cout; out_c += BLOCK_DIM_Z) {
            // printf("Block: (%d, %d, %d), Thread: (%d, %d, %d), out_c: %d\n",
            //         blockIdx.x, blockIdx.y, blockIdx.z,
            //         threadIdx.x, threadIdx.y, threadIdx.z,
            //         out_c);
            T local_sum = 0;
            for (int kx = 0; kx < K; ++kx) { // Kernel rows
                for (int ky = 0; ky < K; ++ky) { // Kernel columns
                    int in_x = out_x + kx;
                    int in_y = out_y + ky;

                    if (in_x < W && in_y < H) {  // Bounds check
                        local_sum += sInput[(threadIdx.y + kx) * (BLOCK_DIM_X + K - 1) + threadIdx.x + ky] *
                                        filter[(out_c * Cin * K * K )+ (in_c * K * K) + (kx * K) + ky];
                    }
                }
            }
            // printf("Block: (%d, %d, %d), Thread: (%d, %d, %d), local sum: %f\n",
            //                 blockIdx.x, blockIdx.y, blockIdx.z,
            //                 threadIdx.x, threadIdx.y, threadIdx.z,
            //                 local_sum);            
            atomicAdd(&result[out_c * H_out * W_out + out_y * W_out + out_x], local_sum);
        }
    }
}

template <typename T>
void launch_conv2d_opt(T *h_result, const T *h_x, const T *h_y, int Cin, int H, int W, int Cout, int K) {
    // Output dimensions
    int H_out = H - K + 1;
    int W_out = W - K + 1;

    // Device memory allocation
    T *d_x, *d_y, *d_result;
    size_t input_size = Cin * H * W * sizeof(T);
    size_t filter_size = Cout * Cin * K * K * sizeof(T);
    size_t output_size = Cout * H_out * W_out * sizeof(T);

    hipMalloc(&d_x, input_size);
    hipMalloc(&d_y, filter_size);
    hipMalloc(&d_result, output_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "1. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Copy inputs to device
    hipMemcpy(d_x, h_x, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, filter_size, hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "2. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z);  
    dim3 gridDim((W_out + blockDim.x - 1) / blockDim.x,
                (H_out + blockDim.y - 1) / blockDim.y,
                Cin);   

    size_t shared_mem_size = (BLOCK_DIM_X + K - 1) * (BLOCK_DIM_Y + K - 1) * sizeof(T);

    // Launch kernel
    conv_kernel_opt<<<gridDim, blockDim, shared_mem_size>>>(d_result, d_x, d_y, Cin, H, W, Cout, K);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "3. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Copy results back to host
    hipMemcpy(h_result, d_result, output_size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
}

template <typename T>
void ref_conv(T *result, const T *input, const T *filter, int Cin, int H, int W, int Cout, int K){
    int H_out = H - K + 1;
    int W_out = W - K + 1;


    for (int cout = 0; cout < Cout; ++cout) {
        for (int h_out = 0; h_out < H_out; ++h_out) {
            for (int w_out = 0; w_out < W_out; ++w_out) {
                T sum = 0;
                for (int cin = 0; cin < Cin; ++cin) {
                    for (int kh = 0; kh < K; ++kh) {
                        for (int kw = 0; kw < K; ++kw) {
                            int input_index = (cin * H * W) 
                                                + ((h_out + kh) * W) 
                                                + (w_out + kw);

                            int filter_index = (cout * Cin * K * K) 
                                                + (cin * K * K) 
                                                + (kh * K) 
                                                + kw;

                            sum += input[input_index] * filter[filter_index];
                        }
                    }
                }
                result[(cout * H_out * W_out) + (h_out * W_out) + w_out] = sum;
            }
        }
    }
    

}

template void launch_conv2d_basic<float>(float *h_result, const float *h_x, const float *h_y, int Cin, int H, int W, int Cout, int K);
template void launch_conv2d_opt<float>(float *h_result, const float *h_x, const float *h_y, int Cin, int H, int W, int Cout, int K);
template void ref_conv<float>(float *h_result, const float *h_x, const float *h_y, int Cin, int H, int W, int Cout, int K);