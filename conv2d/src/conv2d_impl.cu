#include "hip/hip_runtime.h"
#include "conv2d_impl.h"
#include <stdio.h>

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16
#define BLOCK_DIM_Z 2

// Add batch, stride and padding later
// we could have 1 block <-> 1 output channel
// and 1 thread <-> 1 output pixel
template <typename T>
__global__ void conv_kernel_basic(T *result, const T *input, const T *filter, int Cin, int H, int W, int Cout, int K) {
    int x = threadIdx.x;
    int y = threadIdx.y;
    int z = blockIdx.x;

    int H_out = H - K + 1;
    int W_out = W - K + 1;

    T sum = 0;
    if (x < H_out && y < W_out){
        // loop over input channels
        for (int i = 0; i < Cin; i++) {
            // loop over filter dimensions
            for (int j = 0; j < K; j++) {
                for (int k = 0; k < K; k++) {
                    // input of shape (Cin, H, W) with strides (H*W, W, 1)
                    // filter of shape (Cout, Cin, K, K) with strides (Cin*K*K, K*K, K, 1)
                    sum += input[i * H * W + (x + j) * W + (y + k)] * filter[z * Cin * K * K + i * K * K + j * K + k];
                }
            }
        }
        // result of shape (Cout, H_out, W_out) with strides (H_out * W_out, W_out, 1)
        result[z * H_out * W_out + x * W_out + y] = sum;
    }
}

template <typename T>
void launch_conv2d_basic(T *h_result, const T *h_x, const T *h_y, 
                   int Cin, int H, int W, int Cout, int K) {
    // Output dimensions
    int H_out = H - K + 1;
    int W_out = W - K + 1;

    // Device memory allocation
    T *d_x, *d_y, *d_result;
    size_t input_size = Cin * H * W * sizeof(T);
    size_t filter_size = Cout * Cin * K * K * sizeof(T);
    size_t output_size = Cout * H_out * W_out * sizeof(T);

    hipMalloc(&d_x, input_size);
    hipMalloc(&d_y, filter_size);
    hipMalloc(&d_result, output_size);

    // Copy inputs to device
    hipMemcpy(d_x, h_x, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, filter_size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(W_out, H_out); // one thread per output pixel
    dim3 numBlocks(Cout);          // One block per output channel and input channel

    // Launch kernel
    conv_kernel_basic<<<numBlocks, threadsPerBlock>>>(d_result, d_x, d_y, Cin, H, W, Cout, K);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }

    // Copy results back to host
    hipMemcpy(h_result, d_result, output_size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
}

// Supports batching
// Z dim for batch
// X dim for channel
// Y dim for height, width
// template <typename T>
// __global__ void conv_kernel_batched(T *result, const T *input, const T *filter, int N, int Cin, int H, int W, int Cout, int Kh, int Kw) {
//     int x = threadIdx.x; // output channel
//     int y = threadIdx.y; // output pixel
//     int z = threadIdx.z; // batch

//     int H_out = H - Kh + 1;
//     int W_out = W - Kw + 1;

//     int t_r = y / W_out;
//     int t_c = y % W_out;

//     T sum = 0;

//     if (x < Cout && y < H_out * W_out && z < N){
//         for (int i = 0; i < Cin; ++i ){
//             for (int j = 0; j < Kh; ++j){
//                 for (int k = 0; k < Kw; ++k){
//                     printf("z: %d, x: %d, y: %d, i: %d, j: %d, k: %d\n sum += %f * %f", z, x, y, i, j, k, input[(z * Cin * H * W) + (i * H * W) + ((t_r + j) * W) + (k + t_c) ], filter[(x * Cin * Kh * Kw) + (i * Kh * Kw) + (j * Kw) + k ]);
//                     sum += input[(z * Cin * H * W) + (i * H * W) + ((t_r + j) * W) + (k + t_c) ] * filter[(x * Cin * Kh * Kw) + (i * Kh * Kw) + (j * Kw) + k ];
    
//                 }
//             }
//         }

//         // result of shape N, Cout, Hout, Wout
//         result[(z * Cout * H_out * W_out) + (x * H_out * W_out) + (t_r * W_out) + t_c] = sum;

//     }
    
    
// }

template <typename T>
__global__ void conv_kernel_opt(
    T* __restrict__ result, 
    const T* __restrict__ input, 
    const T* __restrict__ filter, 
    int Cin, int H, int W, int Cout, int K) {

    int out_x = threadIdx.x + blockIdx.x * blockDim.x;  // Output width index
    int out_y = threadIdx.y + blockIdx.y * blockDim.y;  // Output height index
    int out_c = blockIdx.z;                            // Output channel index                 

    int H_out = H - K + 1;
    int W_out = W - K + 1;

    // Shared memory for reduction
    __shared__ T partial_sums[BLOCK_DIM_X][BLOCK_DIM_Y];  
    // Initialize shared memory only once per block
    if (threadIdx.z == 0) partial_sums[threadIdx.x][threadIdx.y]= 0.0f;

    __syncthreads();

    T local_sum = 0;
    if (out_x < H_out && out_y < W_out) {
        // Loop over input channels in chunks
        for (int in_c = threadIdx.z; in_c < Cin; in_c += BLOCK_DIM_Z) {
            for (int kx = 0; kx < K; ++kx) { // Kernel rows
                for (int ky = 0; ky < K; ++ky) { // Kernel columns
                    int in_x = out_x + kx;
                    int in_y = out_y + ky;

                    if (in_x < W && in_y < H) {  // Bounds check
                        local_sum += input[in_c * H * W + in_y * W + in_x] *
                                     filter[out_c * Cin * K * K + in_c * K * K + ky * K + kx];
                    }
                }
            }
        }

        // Atomic addition to shared memory
        atomicAdd(&partial_sums[threadIdx.x][threadIdx.y], local_sum);
    }

    __syncthreads();

    // Write final result to global memory
    if (threadIdx.z == 0) {
        if (out_x < H_out && out_y < W_out) {
            result[out_c * H_out * W_out + out_y * W_out + out_x] = partial_sums[threadIdx.x][threadIdx.y];
        }
    }
}


template <typename T>
void launch_conv2d_opt(T *h_result, const T *h_x, const T *h_y, int Cin, int H, int W, int Cout, int K) {
    // Output dimensions
    int H_out = H - K + 1;
    int W_out = W - K + 1;

    // Device memory allocation
    T *d_x, *d_y, *d_result;
    size_t input_size = Cin * H * W * sizeof(T);
    size_t filter_size = Cout * Cin * K * K * sizeof(T);
    size_t output_size = Cout * H_out * W_out * sizeof(T);

    hipMalloc(&d_x, input_size);
    hipMalloc(&d_y, filter_size);
    hipMalloc(&d_result, output_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "1. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Copy inputs to device
    hipMemcpy(d_x, h_x, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, filter_size, hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "2. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Define grid and block dimensions
    dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z);  
    dim3 gridDim((W_out + blockDim.x - 1) / blockDim.x,
                (H_out + blockDim.y - 1) / blockDim.y,
                Cout);   

    // Launch kernel
    conv_kernel_opt<<<gridDim, blockDim>>>(d_result, d_x, d_y, Cin, H, W, Cout, K);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "3. CUDA error: %s\n", hipGetErrorString(err));
    }

    // Copy results back to host
    hipMemcpy(h_result, d_result, output_size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
}

template <typename T>
void ref_conv(T *result, const T *input, const T *filter, int Cin, int H, int W, int Cout, int K){
    int H_out = H - K + 1;
    int W_out = W - K + 1;


    for (int cout = 0; cout < Cout; ++cout) {
        for (int h_out = 0; h_out < H_out; ++h_out) {
            for (int w_out = 0; w_out < W_out; ++w_out) {
                T sum = 0;
                for (int cin = 0; cin < Cin; ++cin) {
                    for (int kh = 0; kh < K; ++kh) {
                        for (int kw = 0; kw < K; ++kw) {
                            int input_index = (cin * H * W) 
                                                + ((h_out + kh) * W) 
                                                + (w_out + kw);

                            int filter_index = (cout * Cin * K * K) 
                                                + (cin * K * K) 
                                                + (kh * K) 
                                                + kw;

                            sum += input[input_index] * filter[filter_index];
                        }
                    }
                }
                result[(cout * H_out * W_out) + (h_out * W_out) + w_out] = sum;
            }
        }
    }
    

}

template void launch_conv2d_basic<float>(float *h_result, const float *h_x, const float *h_y, int Cin, int H, int W, int Cout, int K);
template void launch_conv2d_opt<float>(float *h_result, const float *h_x, const float *h_y, int Cin, int H, int W, int Cout, int K);
template void ref_conv<float>(float *h_result, const float *h_x, const float *h_y, int Cin, int H, int W, int Cout, int K);